
#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdio.h>

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

#define CHECK_LAST_KERN()\
{\
  hipDeviceSynchronize();\
  const hipError_t error=hipGetLastError();\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


struct StreamWraper {
  hipStream_t stream;

  StreamWraper() {
    CHECK( hipStreamCreate(&stream) );
  }

  ~StreamWraper() {
    CHECK( hipStreamDestroy(this->stream) );
  }

  hipStream_t get() {
    return this->stream;
  }
};


extern "C" {
  void* create_stream() {
    StreamWraper* s = new StreamWraper();
    return s;
  }

  void destory_stream(void* stream) {
    StreamWraper* _stream = (StreamWraper*)stream;
    delete _stream;
  }

  void wait_stream(void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    hipStreamSynchronize(s->get());
  }

  void wait_default_stream() {
    hipStreamSynchronize(0);
  }

  void* get_inner_stream(void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
	return (void*)&s->stream;
  }
  
  uint8_t* alloc_locked_buffer(uint32_t bytes) {
    uint8_t* buffer;

    CHECK(hipHostMalloc(&buffer, bytes, hipHostMallocDefault));

    return buffer;
  }

  void free_locked_buffer(uint8_t* buffer) {
    CHECK(hipHostFree(buffer));
  }

  void set_device(size_t index) {
    CHECK(hipSetDevice(index));
  }

  void device_to_host(uint8_t* device, uint8_t* host, uint32_t bytes) {
    CHECK(hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost));
  }

  void device_to_host_with_stream(uint8_t* device, uint8_t* host, uint32_t bytes, void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    CHECK( hipMemcpyAsync(host, device, bytes, hipMemcpyDeviceToHost, s->get()) );
  }


  void host_to_device(uint8_t* host, uint8_t* device, uint32_t bytes) {
    CHECK(hipMemcpy(device, host, bytes, hipMemcpyHostToDevice));
  }

  void host_to_device_with_stream(uint8_t* host, uint8_t* device, uint32_t bytes, void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    CHECK( hipMemcpyAsync(device, host, bytes, hipMemcpyHostToDevice, s->get()) );
  }

  uint8_t* alloc_gpu_buffer(uint32_t bytes) {
    uint8_t* buf;
    CHECK(hipMalloc(&buf, bytes));
    return buf;
  }

  void free_gpu_buffer(uint8_t* buf) {
    CHECK(hipFree(buf));
  }

  void check_and_sync() {
	  CHECK_LAST_KERN();
  }

  uint32_t get_device_count() {
	  int count;
	  hipGetDeviceCount(&count);
	  return (uint32_t)count;
  }

  int32_t get_device_cuda_core_count() {
	  hipDeviceProp_t devProp;
      hipGetDeviceProperties(&devProp, 0);

	  int cores = -1;
	  int mp = devProp.multiProcessorCount;

	  switch (devProp.major){
		  case 2: // Fermi
			  if (devProp.minor == 1) cores = mp * 48;
			  else cores = mp * 32;
			  break;
		  case 3: // Kepler
			  cores = mp * 192;
			  break;
		  case 5: // Maxwell
			  cores = mp * 128;
			  break;
		  case 6: // Pascal
			  if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
			  else if (devProp.minor == 0) cores = mp * 64;
			  break;
		  case 7: // Volta and Turing
			  if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
			  break;
		  case 8: // Ampere
			  if (devProp.minor == 0) cores = mp * 64;
			  else if (devProp.minor == 6) cores = mp * 128;
			  break;
		  default:
			  break;
	  }
	  return cores;
  }

  void cu_memset(uint8_t* devPtr, uint8_t value, size_t count) {
    CHECK(hipMemset((void*)devPtr, value, count));
  }
}

