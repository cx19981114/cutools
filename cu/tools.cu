
#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdio.h>

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

#define CHECK_LAST_KERN()\
{\
  hipDeviceSynchronize();\
  const hipError_t error=hipGetLastError();\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


struct StreamWraper {
  hipStream_t stream;

  StreamWraper() {
    CHECK( hipStreamCreate(&stream) );
  }

  ~StreamWraper() {
    CHECK( hipStreamDestroy(this->stream) );
  }

  hipStream_t get() {
    return this->stream;
  }
};


extern "C" {
  void* create_stream() {
    StreamWraper* s = new StreamWraper();
    return s;
  }

  void destory_stream(void* stream) {
    StreamWraper* _stream = (StreamWraper*)stream;
    delete _stream;
  }

  void wait_stream(void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    hipStreamSynchronize(s->get());
  }

  void wait_default_stream() {
    hipStreamSynchronize(0);
  }

  void* get_inner_stream(void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
	  return (void*)&s->stream;
  }

  uint8_t* alloc_locked_buffer(size_t bytes) {
    uint8_t* buffer;
    CHECK(hipHostMalloc(&buffer, bytes, hipHostMallocDefault));
    return buffer;
  }

  void free_locked_buffer(uint8_t* buffer) {
    CHECK(hipHostFree(buffer));
  }

  void set_device(size_t index) {
    CHECK(hipSetDevice(index));
  }

  void set_device_flags(uint32_t flags) {
    CHECK(hipSetDeviceFlags(flags))
  }

  void device_to_host(uint8_t* device, uint8_t* host, size_t bytes) {
    CHECK(hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost));
  }

  void host_to_device(uint8_t* host, uint8_t* device, size_t bytes) {
    CHECK(hipMemcpy(device, host, bytes, hipMemcpyHostToDevice));
  }

  void host_to_device_2d(uint8_t* host, uint8_t* device,
                         size_t hostPitch, size_t devPitch,
                         size_t width, size_t height) {
    CHECK(hipMemcpy2D(device, devPitch, host, hostPitch, width, height, hipMemcpyHostToDevice));
  }

  uint8_t* alloc_gpu_buffer(size_t bytes) {
    uint8_t* buf;
    CHECK(hipMalloc(&buf, bytes));
    return buf;
  }

  void free_gpu_buffer(uint8_t* buf) {
    CHECK(hipFree(buf));
  }

  void check_and_sync() {
	  CHECK_LAST_KERN();
  }

  void sync_device() {
    CHECK(hipDeviceSynchronize());
  }

  void reset_device() {
    CHECK(hipDeviceReset());
  }

  uint32_t get_last_error() {
    const hipError_t error=hipGetLastError();
    return (uint32_t)(error);
  }

  const char* get_error_string(uint32_t error_code) {
    return hipGetErrorString((hipError_t)error_code);
  }

  uint32_t get_device_count() {
	  int count;
	  hipGetDeviceCount(&count);
	  return (uint32_t)count;
  }

  uint32_t get_sm_count() {
	  hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
	  return devProp.multiProcessorCount;
  }

  uint32_t get_device_cuda_core_count() {
	  uint32_t cores = 0;

	  hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    uint32_t mp = devProp.multiProcessorCount;

	  switch (devProp.major){
		  case 2: // Fermi
			  if (devProp.minor == 1) cores = mp * 48;
			  else cores = mp * 32;
			  break;
		  case 3: // Kepler
			  cores = mp * 192;
			  break;
		  case 5: // Maxwell
			  cores = mp * 128;
			  break;
		  case 6: // Pascal
			  if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
			  else if (devProp.minor == 0) cores = mp * 64;
			  break;
		  case 7: // Volta and Turing
			  if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
			  break;
		  case 8: // Ampere
			  if (devProp.minor == 0) cores = mp * 64;
			  else if (devProp.minor == 6) cores = mp * 128;
			  break;
		  default:
			  break;
	  }
	  return cores;
  }

  void cu_memset(uint8_t* devPtr, uint8_t value, size_t count) {
    CHECK(hipMemset((void*)devPtr, value, count));
  }
}

