
#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdio.h>

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

#define CHECK_LAST_KERN()\
{\
  hipDeviceSynchronize();\
  const hipError_t error=hipGetLastError();\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


struct StreamWraper {
  hipStream_t stream;

  StreamWraper() {
    CHECK( hipStreamCreate(&stream) );
  }

  ~StreamWraper() {
    CHECK( hipStreamDestroy(this->stream) );
  }

  hipStream_t get() {
    return this->stream;
  }
};


extern "C" {
  void* create_stream() {
    StreamWraper* s = new StreamWraper();
    return s;
  }

  void destory_stream(void* stream) {
    StreamWraper* _stream = (StreamWraper*)stream;
    delete _stream;
  }

  void wait_stream(void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    hipStreamSynchronize(s->get());
  }

  void* get_inner_stream(void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
	return (void*)&s->stream;
  }
  
  uint8_t* alloc_locked_buffer(uint32_t bytes) {
    uint8_t* buffer;

    CHECK(hipHostMalloc(&buffer, bytes, hipHostMallocDefault));

    return buffer;
  }

  void free_locked_buffer(uint8_t* buffer) {
    CHECK(hipHostFree(buffer));
  }

  void set_device(size_t index) {
    CHECK(hipSetDevice(index));
  }

  void device_to_host(uint8_t* device, uint8_t* host, uint32_t bytes) {
    CHECK(hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost));
  }

  void device_to_host_with_stream(uint8_t* device, uint8_t* host, uint32_t bytes, void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    CHECK( hipMemcpyAsync(host, device, bytes, hipMemcpyDeviceToHost, s->get()) );
  }


  void host_to_device(uint8_t* host, uint8_t* device, uint32_t bytes) {
    CHECK(hipMemcpy(device, host, bytes, hipMemcpyHostToDevice));
  }

  void host_to_device_with_stream(uint8_t* host, uint8_t* device, uint32_t bytes, void* stream) {
    StreamWraper* s = (StreamWraper*)stream;
    CHECK( hipMemcpyAsync(device, host, bytes, hipMemcpyHostToDevice, s->get()) );
  }

  uint8_t* alloc_gpu_buffer(uint32_t bytes) {
    uint8_t* buf;
    CHECK(hipMalloc(&buf, bytes));
    return buf;
  }

  void free_gpu_buffer(uint8_t* buf) {
    CHECK(hipFree(buf));
  }
}

